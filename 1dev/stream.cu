#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <numeric>
#include <chrono>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "\tError: %s:%d, ", __FILE__, __LINE__);               \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

void d2d_copy(float* d_from, float* d_to, u_int64_t N, hipStream_t stream, int ite = 1);
void d2d_scale(float* d_from, float* d_to, u_int64_t N, hipStream_t stream, int ite = 1);
void d2d_sum(float* d_from1, float* d_from2, float* d_to, u_int64_t N, hipStream_t stream, int ite = 1);
void d2d_triad(float* d_from1, float* d_from2, float* d_to, u_int64_t N, hipStream_t stream, int ite = 1);

__global__ void copy_gpu(float* d_from, float* d_to, u_int64_t N){
    u_int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)d_to[idx] = d_from[idx];
}

__global__ void scale_gpu(float* d_from, float* d_to, u_int64_t N){
    u_int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)d_to[idx] = 9 * d_from[idx];
}

__global__ void sum_gpu(float* d_from1, float* d_from2, float* d_to, u_int64_t N){
    u_int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)d_to[idx] = d_from1[idx] + d_from2[idx];
}

__global__ void triad_gpu(float* d_from1, float* d_from2, float* d_to, u_int64_t N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)d_to[idx] = d_from1[idx] + 9 * d_from2[idx];
}

void printtime(hipEvent_t start[], hipEvent_t end[], u_int64_t N, int ite);
void enableP2P();

int main(int argc, char const* argv[]){

    u_int64_t N = 1 << 30; //1G
    // N = N * 4;
    u_int64_t N2 = N; //1<<25;
    float* d_dataA, * d_dataB, * d_dataC, * d_data1D;       //4B
    int iteration = 5;
    CHECK(hipMalloc(&d_dataA, N * sizeof(float)));
    CHECK(hipMalloc(&d_dataB, N * sizeof(float)));
    CHECK(hipMalloc(&d_dataC, N * sizeof(float)));


    float* h_data = new float[N];
    for (u_int64_t i = 0; i < N; ++i)h_data[i] = rand();
    std::vector<std::chrono::high_resolution_clock::time_point> cpytime(5);
    cpytime[0] = std::chrono::high_resolution_clock::now();
    CHECK(hipMemcpy(d_dataA, h_data, N * sizeof(float), hipMemcpyHostToDevice));
    cpytime[1] = std::chrono::high_resolution_clock::now();
    CHECK(hipMemcpy(d_dataB, h_data, N * sizeof(float), hipMemcpyHostToDevice));
    cpytime[2] = std::chrono::high_resolution_clock::now();
    CHECK(hipMemcpy(d_dataC, h_data, N * sizeof(float), hipMemcpyHostToDevice));
    cpytime[3] = std::chrono::high_resolution_clock::now();

    std::vector<float> cpyt1(4);
    for (u_int64_t i = 0; i < 4; i++){
        cpyt1[i] = std::chrono::duration<float, std::milli>(cpytime[i+1] - cpytime[i]).count();
        std::cout << "\t" << i << "th Time: " << cpyt1[i] << "ms" << std::endl;
    }

    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));


    std::cout << "copy" << std::endl;
    for (uint64_t i = N2; i < N + 1; i <<= 1){
        d2d_copy(d_dataC, d_dataA, i, stream, iteration);
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "scale" << std::endl;
    for (uint64_t i = N2; i < N + 1; i <<= 1){
        d2d_scale(d_dataC, d_dataA, i, stream, iteration);
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "sum" << std::endl;
    for (uint64_t i = N2; i < N + 1; i <<= 1){
        d2d_sum(d_dataC, d_data1D, d_dataA, i, stream, iteration);
        CHECK(hipDeviceSynchronize());
    }

    std::cout << "triad" << std::endl;
    for (uint64_t i = N2; i < N + 1; i <<= 1){
        d2d_triad(d_dataC, d_data1D, d_dataA, i, stream, iteration);
        CHECK(hipDeviceSynchronize());
    }

    delete[] h_data;
    CHECK(hipFree(d_dataA));CHECK(hipFree(d_dataC));CHECK(hipFree(d_data1D));
    CHECK(hipStreamDestroy(stream));

    return 0;
}

void d2d_copy(float* d_from, float* d_to, u_int64_t N, hipStream_t stream, int ite){
    hipEvent_t start[ite], end[ite];
    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventCreate(&start[i]));
        CHECK(hipEventCreate(&end[i]));
    }

    dim3 block(1024, 1, 1);
    dim3 grid((N + block.x - 1) / block.x, 1, 1);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventRecord(start[i], stream));
        copy_gpu <<<grid, block, 0, stream>>> (d_from, d_to, N);
        CHECK(hipEventRecord(end[i], stream));
    }
    CHECK(hipDeviceSynchronize());

    printtime(start, end, N, ite);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventDestroy(start[i]));
        CHECK(hipEventDestroy(end[i]));
    }
}

void d2d_scale(float* d_from, float* d_to, u_int64_t N, hipStream_t stream, int ite){
    hipEvent_t start[ite], end[ite];
    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventCreate(&start[i]));
        CHECK(hipEventCreate(&end[i]));
    }

    dim3 block(1024, 1, 1);
    dim3 grid((N + block.x - 1) / block.x, 1, 1);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventRecord(start[i], stream));
        scale_gpu <<<grid, block, 0, stream>>> (d_from, d_to, N);
        CHECK(hipEventRecord(end[i], stream));
    }
    CHECK(hipDeviceSynchronize());

    printtime(start, end, N, ite);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventDestroy(start[i]));
        CHECK(hipEventDestroy(end[i]));
    }
}

void d2d_sum(float* d_from1,float* d_from2, float* d_to, u_int64_t N, hipStream_t stream, int ite){
    hipEvent_t start[ite], end[ite];
    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventCreate(&start[i]));
        CHECK(hipEventCreate(&end[i]));
    }

    dim3 block(1024, 1, 1);
    dim3 grid((N + block.x - 1) / block.x, 1, 1);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventRecord(start[i], stream));
        sum_gpu <<<grid, block, 0, stream>>> (d_from1, d_from2, d_to, N);
        CHECK(hipEventRecord(end[i], stream));
    }
    CHECK(hipDeviceSynchronize());

    printtime(start, end, N, ite);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventDestroy(start[i]));
        CHECK(hipEventDestroy(end[i]));
    }
}

void d2d_triad(float* d_from1, float* d_from2, float* d_to, u_int64_t N, hipStream_t stream, int ite){
    hipEvent_t start[ite], end[ite];
    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventCreate(&start[i]));
        CHECK(hipEventCreate(&end[i]));
    }

    dim3 block(1024, 1, 1);
    dim3 grid((N + block.x - 1) / block.x, 1, 1);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventRecord(start[i], stream));
        triad_gpu <<<grid, block, 0, stream>>> (d_from1, d_from2, d_to, N);
        CHECK(hipEventRecord(end[i], stream));
    }
    CHECK(hipDeviceSynchronize());

    printtime(start, end, N, ite);

    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventDestroy(start[i]));
        CHECK(hipEventDestroy(end[i]));
    }
}

void printtime(hipEvent_t start[], hipEvent_t end[], u_int64_t N, int ite){
    std::vector<float> t1(ite);
    for (u_int64_t i = 0; i < ite; i++){
        CHECK(hipEventElapsedTime(&t1[i], start[i], end[i]));
    }
    float sum = std::accumulate(t1.begin() + 1, t1.end(), 0.0f);
    float mean_time = sum / (ite - 1);
    std::cout << " N: " << (N>>20) << "M iter: " << ite << std::endl;
    std::cout << "\t1st Time: " << t1[0] << "ms" << std::endl;
    std::cout << "\tMean Time: " << mean_time << "ms" << std::endl;
    std::cout << "\tBandwidth: " << (N * sizeof(float) >> 20) / (mean_time / 1000) << "MB/s (" << ((N * sizeof(float) >> 20) / (mean_time / 1000)) / (1 << 10) << "GB/s)" << std::endl;
}
