#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "\tError: %s:%d, ", __FILE__, __LINE__);               \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

void h2d_memcpy(float* d_data, float* h_data, u_int64_t N, hipStream_t stream,int ite = 1);
void enableP2P();

int main(){

    u_int64_t N = 1 << 30; //1G
    float* d_data;       //4B
    CHECK(hipMalloc(&d_data, N * sizeof(float)));

    float* h_data = new float[N];
    for (int i = 0; i < N; ++i)h_data[i] = rand();

    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));

    h2d_memcpy(d_data, h_data, N, stream,2);

    CHECK(hipDeviceSynchronize());



    
    delete[] h_data;
    CHECK(hipFree(d_data));


    return 0;
}

void h2d_memcpy(float* d_data, float* h_data, u_int64_t N, hipStream_t stream,int ite){
    int tNum = 10;
    hipEvent_t time[ite*tNum];
    for (int i = 0; i < ite*tNum; i++) CHECK(hipEventCreate(&time[i]));
    float* h_result = new float[N];

    for (size_t k = 0; k < ite; k++){
        //h2d
        CHECK(hipEventRecord(time[k*ite+0], stream));
        CHECK(hipMemcpyAsync(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice, stream));
        CHECK(hipEventRecord(time[k*ite+1], stream));
        CHECK(hipEventSynchronize(time[k*ite+1]));

        // ちゃんと転送されていることを確認するための操作
        thrust::device_ptr<float> d_vec(d_data);
        thrust::sort(thrust::cuda::par.on(stream), d_vec, d_vec + N);
        CHECK(hipEventRecord(time[k*ite+2], stream));
        CHECK(hipEventSynchronize(time[k*ite+2]));

        //d2h
        CHECK(hipEventRecord(time[k*ite+3], stream));
        CHECK(hipMemcpyAsync(h_result, d_data, N * sizeof(float), hipMemcpyDeviceToHost, stream));
        CHECK(hipEventRecord(time[k*ite+4], stream));
        CHECK(hipEventSynchronize(time[k*ite+4]));

        std::cout << "Sorted :";
        for (u_int64_t i = 0; i < 10; i++){
            std::cout << h_result[i] << " ";
        }
        std::cout << std::endl;

    }
    
    for (size_t k = 0; k < ite; k++){
        float mstime = 0;
        CHECK(hipEventElapsedTime(&mstime, time[k*ite+0], time[k*ite+1]));
        std::cout << "time: " << mstime << " ms" << std::endl;
        CHECK(hipEventElapsedTime(&mstime, time[k*ite+3], time[k*ite+4]));
        std::cout << "time: " << mstime << " ms" << std::endl;
    }
    
    
    CHECK(hipStreamDestroy(stream));
    for (int i = 0; i < tNum; i++) CHECK(hipEventDestroy(time[i]));
    CHECK(hipDeviceSynchronize());
}

void enableP2P(){
    int ndev;
    CHECK(hipGetDeviceCount(&ndev));
    for (int dev_id = 0; dev_id < ndev; dev_id++){//P2Pのデバイス間での有効化
        CHECK(hipSetDevice(dev_id));
        for (int j = 0; j < ndev; j++){
            if (dev_id == j) continue;
            int peer_access_available = 0;
            CHECK(hipDeviceCanAccessPeer(&peer_access_available, dev_id, j));

            if (peer_access_available){
                CHECK(hipDeviceEnablePeerAccess(j, 0));
                printf("> GPU%d enabled direct access to GPU%d\n", dev_id, j);
            } else{
                printf("(%d, %d)\n", dev_id, j);
            }
        }
    }
}